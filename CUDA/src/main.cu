#include "hip/hip_runtime.h"
﻿#include "../common/book.h"
#include "../common/cpu_bitmap.h"

#include "hip/hip_runtime.h"
#include ""

#define DIM 1000

struct hipComplex
{
    float r;
    float i;
    // note: here we need to declare ctor as __device__ as well
    __device__ hipComplex(float a, float b) : r(a), i(b) {}
    __device__ float magnitude2(void)
    {
        return r * r + i * i;
    }
    __device__ hipComplex operator*(const hipComplex &a)
    {
        return hipComplex(r * a.r - i * a.i, i * a.r + r * a.i);
    }
    __device__ hipComplex operator+(const hipComplex &a)
    {
        return hipComplex(r + a.r, i + a.i);
    }
};

__device__ int julia(int x, int y)
{
    const float scale = 1.5;
    float jx = scale * (float)(DIM / 2 - x) / (DIM / 2);
    float jy = scale * (float)(DIM / 2 - y) / (DIM / 2);

    hipComplex c(-0.8, 0.156);
    hipComplex a(jx, jy);

    int i = 0;
    for (i = 0; i < 200; i++)
    {
        a = a * a + c;
        if (a.magnitude2() > 1000)
            return 0;
    }

    return 1;
}

__global__ void kernel(unsigned char *ptr)
{
    // Note: unlike in CPU version, we don't have for loop to generate pixel indices anymore/
    // in GPU, we compute pixel indices through blockIdx

    // map from blockIdx to pixel position
    // (x,y) ranges from (0, 0) and (DIM - 1, DIM - 1)
    int x = blockIdx.x;
    int y = blockIdx.y;
    // offset ranges from 0 to (DIM * DIM - 1)
    int offset = x + y * gridDim.x;

    // now calculate the value at that position
    int juliaValue = julia(x, y);
    ptr[offset * 4 + 0] = 255 * juliaValue;
    ptr[offset * 4 + 1] = 0;
    ptr[offset * 4 + 2] = 0;
    ptr[offset * 4 + 3] = 255;
}

int main(void)
{
    CPUBitmap bitmap(DIM, DIM);
    // declare a pointer to hold a copy of the data on the device
    unsigned char *dev_bitmap;

    HANDLE_ERROR(hipMalloc((void **)&dev_bitmap, bitmap.image_size()));

    // specify 2D grid of blocks because our problem is 2D
    dim3 grid(DIM, DIM);
    kernel<<<grid, 1>>>(dev_bitmap);

    HANDLE_ERROR(hipMemcpy(bitmap.get_ptr(), dev_bitmap,
                            bitmap.image_size(),
                            hipMemcpyDeviceToHost));

    HANDLE_ERROR(hipFree(dev_bitmap));

    bitmap.display_and_exit();
}
